#include "hip/hip_runtime.h"
#include <cstdio>
#include <err.h>
#include <fstream>
#include <raft/core/device_resources.hpp>
#include <raft/core/device_span.hpp>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/mr/device/cuda_async_memory_resource.hpp>
#include <rmm/mr/device/owning_wrapper.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <sstream>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <vector>

static auto make_async()
{
    return std::make_shared<rmm::mr::cuda_async_memory_resource>();
}

static auto make_pool()
{
    // Allocate 0.05 Go
    size_t initial_pool_size = std::pow(2, 26);
    return rmm::mr::make_owning_wrapper<rmm::mr::pool_memory_resource>(
        make_async(), initial_pool_size);
}

static unsigned long long q1(raft::device_resources& res,
                             const std::string& filename)
{
    // reading the input
    std::vector<unsigned long long> left;
    std::vector<unsigned long long> right;
    std::ifstream input(filename);
    std::string line;
    while (getline(input, line))
    {
        // Inserting in a sorted manner
        std::stringstream ss(line);
        unsigned long long first = 0;
        ss >> first;
        auto lower = std::lower_bound(left.begin(), left.end(), first);
        left.insert(lower, first);
        ss >> first;
        lower = std::lower_bound(right.begin(), right.end(), first);
        right.insert(lower, first);
    }

    // Putting the data on GPU
    rmm::device_uvector<unsigned long long> dleft(left.size(),
                                                  res.get_stream());
    raft::copy(dleft.data(), left.data(), left.size(), dleft.stream());
    rmm::device_uvector<unsigned long long> dright(right.size(),
                                                   dleft.stream());
    raft::copy(dright.data(), right.data(), right.size(), dright.stream());

    // Creating some fancy iterators
    // Zipping both vectors
    const auto q1_iterator = thrust::make_zip_iterator(
        thrust::make_tuple(dleft.cbegin(), dright.cbegin()));
    // Preparing the distance computation using a transform iterator
    const auto distance = thrust::make_transform_iterator(
        q1_iterator,
        [] __device__(
            thrust::tuple<unsigned long long, unsigned long long> const& tpl)
            -> unsigned long long {
            return thrust::get<0>(tpl) > thrust::get<1>(tpl)
                ? thrust::get<0>(tpl) - thrust::get<1>(tpl)
                : thrust::get<1>(tpl) - thrust::get<0>(tpl);
        });
    unsigned long long init = 0;
    // Computing the reduce
    auto result = thrust::reduce(thrust::cuda::par.on(dleft.stream()), distance,
                                 distance + left.size(), init);
    return result;
}

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        errx(1, "Error: please provide a filename");
    }

    // RMM Setup
    auto memory_resource = make_pool();
    rmm::mr::set_current_device_resource(memory_resource.get());
    //
    // Raft Setup
    raft::device_resources res;

    unsigned long long q1_res = q1(res, argv[1]);

    std::cout << "q1: " << q1_res << "\n";

    unsigned long long q2_res = 0;

    std::cout << "q2: " << q2_res << std::endl;

    return 0;
}

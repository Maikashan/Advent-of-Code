#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <err.h>
#include <fstream>
#include <raft/core/device_resources.hpp>
#include <raft/core/device_span.hpp>
#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/cuda_async_memory_resource.hpp>
#include <rmm/mr/device/owning_wrapper.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <sstream>
#include <vector>

#include "rmm/cuda_stream_view.hpp"

#define CUDA_CHECK_ERROR(call)                                                 \
    do                                                                         \
    {                                                                          \
        hipError_t err = call;                                                \
        if (err != hipSuccess)                                                \
        {                                                                      \
            std::cerr << "CUDA error in " << __FILE__ << " at line "           \
                      << __LINE__ << ": " << hipGetErrorString(err)           \
                      << std::endl;                                            \
            std::exit(EXIT_FAILURE);                                           \
        }                                                                      \
    } while (0)

static auto make_async()
{
    return std::make_shared<rmm::mr::cuda_async_memory_resource>();
}

static auto make_pool()
{
    // Allocate 0.05 Go
    size_t initial_pool_size = std::pow(2, 26);
    return rmm::mr::make_owning_wrapper<rmm::mr::pool_memory_resource>(
        make_async(), initial_pool_size);
}

__inline__ __device__ int warp_reduce(int val)
{
#pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(~0, val, offset);
    return val;
}

__global__ void level_handling_block(raft::device_span<int> data,
                                     raft::device_span<int> result)
{
    extern __shared__ int sdata[];
    // The second part of the shared memory is allocated for the flags
    int* flags = (sdata + blockDim.x);

    unsigned int tid = threadIdx.x;
    // We should never have more than a block
    unsigned int i = blockDim.x * blockIdx.x + tid;
    if (i >= data.size())
        return;

    sdata[tid] = data[tid];
    __syncthreads();

    flags[tid] = tid != 0 && sdata[tid] > sdata[tid - 1] ? 1 : 0;
    __syncthreads();

    // Block level reduce, without one value (the first in reality)
    for (int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s && i + s < data.size())
            flags[tid] += flags[tid + s];
        __syncthreads();
    }

    int val = 0;

    if (tid < 32)
        val = warp_reduce(flags[tid]);

    if (tid == 0)
        flags[0] = val;

    __syncthreads();

    // The reduced value must be equal to 0 or the size - 1 if the data is
    // sorted
    if (flags[0] != 0 && flags[0] < data.size() - 1)
    {
        return;
    }
    __syncthreads();

    // Checking if the distance if in the said boundaries
    if (tid > 0)
    {
        int dist = abs(sdata[tid] - sdata[tid - 1]);
        flags[tid] = dist >= 1 && dist <= 3 ? 1 : 0;
    }
    else
    {
        flags[tid] = 0;
    }
    __syncthreads();

    // Block level Reduce (once again)
    for (int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s && i + s < data.size() - 1)
            flags[tid] += flags[tid + s];
        __syncthreads();
    }

    val = 0;
    if (tid < 32)
        val = warp_reduce(flags[tid]);

    if (tid == 0 && val == data.size() - 1)
        atomicAdd(&result[0], 1);
}

static int part_1(const std::string& filename)
{
    // reading the input
    std::ifstream input(filename);
    std::string line;
    int* result;
    CUDA_CHECK_ERROR(hipMalloc(&result, sizeof(int)));
    CUDA_CHECK_ERROR(hipMemset(result, 0, sizeof(int)));

    CUDA_CHECK_ERROR(hipStreamSynchronize(0));

    // If every line was of the same size, we could use MPI to read each line
    // with fseek and make a better use of the streams
    std::vector<hipStream_t> streams;
    while (getline(input, line))
    {
        hipStream_t s;
        CUDA_CHECK_ERROR(hipStreamCreate(&s));

        // If we were not using GPU, we could handle most of the logic here
        std::vector<int> level;
        std::stringstream ss(line);
        int first = 0;
        while (ss >> first)
        {
            level.push_back(first);
        }
        // Putting every line on GPU
        rmm::device_uvector<int> dlevel(level.size(), s);
        raft::copy(dlevel.data(), level.data(), level.size(), s);
        streams.emplace_back(s);

        // I have multiple issue with this code.
        // - The rows do not all have the same amount of columns -> Challenging
        // to efficiently prepare my data for the GPU, and will have a lot of
        // work imbalance.
        // - Each row is very short, not very worth it to use my gpu on one line
        // Since it is only the second day and i am late, i will do something
        // neither optimized nor smart

        // We assume the length of a line will never be more than 1024 (maximum
        // size for a block)
        level_handling_block<<<1, level.size(),
                               dlevel.size() * 2 * sizeof(int) - 1, s>>>(
            raft::device_span<int>(dlevel.data(), dlevel.size()),
            raft::device_span<int>(result, 1));
    }

    for (hipStream_t& s : streams)
    {
        CUDA_CHECK_ERROR(hipStreamSynchronize(s));
        CUDA_CHECK_ERROR(hipStreamDestroy(s));
    }

    int final_result;
    CUDA_CHECK_ERROR(
        hipMemcpy(&final_result, result, sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK_ERROR(hipFree(result));

    return final_result;
}

__global__ void level_handling_block_unified(raft::device_span<int> data,
                                             raft::device_span<int> result)
{
    extern __shared__ int sdata[];
    // The second part of the shared memory is allocated for the flags
    int* flags = (sdata + blockDim.x);

    unsigned int tid = threadIdx.x;
    // We should never have more than a block
    unsigned int i = blockDim.x * blockIdx.x + tid;
    if (i >= data.size())
        return;

    sdata[tid] = data[tid];
    __syncthreads();

    flags[tid] = tid != 0 && sdata[tid] > sdata[tid - 1] ? 1 : 0;
    __syncthreads();

    // Block level reduce, without one value (the first in reality)
    for (int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s && i + s < data.size())
            flags[tid] += flags[tid + s];
        __syncthreads();
    }

    int val = 0;

    if (tid < 32)
        val = warp_reduce(flags[tid]);

    if (tid == 0)
        flags[0] = val;

    __syncthreads();

    // The reduced value must be equal to 0 or the size - 1 if the data is
    // sorted
    if (flags[0] != 0 && flags[0] < data.size() - 1)
    {
        return;
    }
    __syncthreads();

    // Checking if the distance if in the said boundaries
    if (tid > 0)
    {
        int dist = abs(sdata[tid] - sdata[tid - 1]);
        flags[tid] = dist >= 1 && dist <= 3 ? 1 : 0;
    }
    else
    {
        flags[tid] = 0;
    }
    __syncthreads();

    // Block level Reduce (once again)
    for (int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s && i + s < data.size() - 1)
            flags[tid] += flags[tid + s];
        __syncthreads();
    }

    val = 0;
    if (tid < 32)
        val = warp_reduce(flags[tid]);

    if (tid == 0 && val == data.size() - 1)
        result[0]++;
}

static int part_2(const std::string& filename)
{
    std::ifstream input(filename);
    std::string line;
    int tot_result = 0;

    // If every line was of the same size, we could use MPI to read each line
    // with fseek and make a better use of the streams
    while (getline(input, line))
    {
        // If we were not using GPU, we could handle most of the logic here
        std::vector<int> level;
        std::stringstream ss(line);
        int first = 0;
        while (ss >> first)
        {
            level.push_back(first);
        }
        int* result;
        CUDA_CHECK_ERROR(hipMallocManaged(&result, sizeof(int)));
        CUDA_CHECK_ERROR(hipMemset(result, 0, sizeof(int)));

        {
            rmm::device_uvector<int> dlevel(level.size(),
                                            rmm::cuda_stream_default);
            raft::copy(dlevel.data(), level.data(), level.size(),
                       rmm::cuda_stream_default);

            level_handling_block<<<1, level.size(),
                                   dlevel.size() * 2 * sizeof(int) - 1>>>(
                raft::device_span<int>(dlevel.data(), dlevel.size()),
                raft::device_span<int>(result, 1));
            hipDeviceSynchronize();
        }

        // Well... bruteforce is a way...
        size_t i = 0;
        while (*result == 0 && i < level.size())
        {
            std::vector<int> short_level;
            for (int j = 0; j < level.size(); j++)
            {
                if (j == i)
                    continue;
                short_level.push_back(level[j]);
            }
            rmm::device_uvector<int> dlevel(short_level.size(),
                                            rmm::cuda_stream_default);
            raft::copy(dlevel.data(), short_level.data(), short_level.size(),
                       rmm::cuda_stream_default);

            level_handling_block_unified<<<
                1, short_level.size(), dlevel.size() * 2 * sizeof(int) - 1>>>(
                raft::device_span<int>(dlevel.data(), dlevel.size()),
                raft::device_span<int>(result, 1));
            hipDeviceSynchronize();
            i++;
        }
        tot_result += *result;
        CUDA_CHECK_ERROR(hipFree(result));
    }

    return tot_result;
}

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        errx(1, "Error: please provide a filename");
    }

    // RMM Setup
    auto memory_resource = make_pool();
    rmm::mr::set_current_device_resource(memory_resource.get());

    unsigned long long part1_res = part_1(argv[1]);

    std::cout << "q1: " << part1_res << "\n";

    unsigned long long part2_res = part_2(argv[1]);

    std::cout << "q2: " << part2_res << std::endl;

    return 0;
}
